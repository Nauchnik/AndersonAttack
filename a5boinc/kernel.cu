#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime_api.h>


#define R1LEN 19
#define R2LEN 22
#define R3LEN 23

#define R2MAJ 11

#define BLOCKSIZE 8


#define m_CopyToGPU(d, s,  bytes){ checkCudaErrors(hipMalloc((void**) &d, bytes)); checkCudaErrors(hipMemcpy((void*) d, (void*)s, bytes, hipMemcpyHostToDevice)); }
typedef unsigned regtype;


inline void CUDA_CHECK_ERROR2(hipError_t err)
{
	if (err != hipSuccess) { printf("Cuda error %i: %s\n ", (int)err, hipGetErrorString(err)); printf("Error in file: %s, line: %i\n", __FILE__, __LINE__); }
}

static inline double cpuTime(void) { return (double)clock() / CLOCKS_PER_SEC; }

const unsigned KSTREAMLEN = 64; // ����� ��������� ������ � ������
//char stream[KSTREAMLEN] = { 2, 3, 2, 4, 3, 6, 1, 1, 8, 10,2, 2, 2, 4, 4, 6, 2, 8, 9, 10,1, 2, 3, 4, 5, 6, 7, 8, 9, 10,2, 2, 3, 4, 5, 7, 7, 8, 2, 10,1, 2, 3, 4, 5, 6, 7, 8, 9, 10,1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 1,2,3,4 };

__constant__ char stream_gpu[KSTREAMLEN] = { 2, 3, 2, 4, 3, 6, 1, 1, 8, 10,2, 2, 2, 4, 4, 6, 2, 8, 9, 10,1, 2, 3, 4, 5, 6, 7, 8, 9, 10,2, 2, 3, 4, 5, 7, 7, 8, 2, 10,1, 2, 3, 4, 5, 6, 7, 8, 9, 10,1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 1,2,3,4 };

inline unsigned expand_cpu(bool b) { return (0 - b); }
__device__ inline unsigned expand(bool b) { return (0 - b); }
__device__ unsigned   gpu_result = 0;

#define DEF_REGS_R1 regtype& b01, regtype& b02, regtype& b03, regtype& b04, regtype& b05, regtype& b06, regtype& b07, regtype& b08, regtype& b09, regtype& b10, regtype& b11, regtype& b12, regtype& b13, regtype& b14, regtype& b15, regtype& b16, regtype& b17, regtype& b18, regtype& b19

#define DEF_REGS_R2 regtype& b01, regtype& b02, regtype& b03, regtype& b04, regtype& b05, regtype& b06, regtype& b07, regtype& b08, regtype& b09, regtype& b10, regtype& b11, regtype& b12, regtype& b13, regtype& b14, regtype& b15, regtype& b16, regtype& b17, regtype& b18, regtype& b19, regtype& b20, regtype& b21, regtype& b22

#define DEF_REGS_R3 regtype& b01, regtype& b02, regtype& b03, regtype& b04, regtype& b05, regtype& b06, regtype& b07, regtype& b08, regtype& b09, regtype& b10, regtype& b11, regtype& b12, regtype& b13, regtype& b14, regtype& b15, regtype& b16, regtype& b17, regtype& b18, regtype& b19, regtype& b20, regtype& b21, regtype& b22, regtype& b23

#define REGS_R1 r1b01, r1b02, r1b03, r1b04, r1b05, r1b06, r1b07, r1b08, r1b09, r1b10, r1b11, r1b12, r1b13, r1b14, r1b15, r1b16, r1b17, r1b18, r1b19
#define REGS_R2	r2b01, r2b02, r2b03, r2b04, r2b05, r2b06, r2b07, r2b08, r2b09, r2b10, r2b11, r2b12, r2b13, r2b14, r2b15, r2b16, r2b17, r2b18, r2b19, r2b20, r2b21, r2b22
#define REGS_R3	r3b01, r3b02, r3b03, r3b04, r3b05, r3b06, r3b07, r3b08, r3b09, r3b10, r3b11, r3b12, r3b13, r3b14, r3b15, r3b16, r3b17, r3b18, r3b19, r3b20, r3b21, r3b22, r3b23




#define  DEF_R1(n) regtype \
r1b01=expand(1&(n>>0)),\
r1b02=expand(1&(n>>1)),\
r1b03=expand(1&(n>>2)),\
r1b04=expand(1&(n>>3)),\
r1b05=expand(1&(n>>4)),\
r1b06=expand(1&(n>>5)),\
r1b07=expand(1&(n>>6)),\
r1b08=expand(1&(n>>7)),\
r1b09=expand(1&(n>>8)),\
r1b10=expand(1&(n>>9)),\
r1b11=expand(1&(n>>10)),\
r1b12=expand(1&(n>>11)),\
r1b13=expand(1&(n>>12)),\
r1b14=expand(1&(n>>13)),\
r1b15=expand(1&(n>>14)),\
r1b16=expand(1&(n>>15)),\
r1b17=expand(1&(n>>16)),\
r1b18=expand(1&(n>>17)),\
r1b19=expand(1&(n>>18))



#define DEF_R2(n) regtype \
r2b01=0xffff,\
r2b02=0xff00ff,\
r2b03=0xf0f0f0f0,\
r2b04=0x33333333,\
r2b05=0x55555555,\
r2b06=expand(1&(n>>5)),\
r2b07=expand(1&(n>>6)),\
r2b08=expand(1&(n>>7)),\
r2b09=expand(1&(n>>8)),\
r2b10=expand(1&(n>>9)),\
r2b11=expand(1&(n>>10)),\
r2b12=0,\
r2b13=0,\
r2b14=0,\
r2b15=0,\
r2b16=0,\
r2b17=0,\
r2b18=0,\
r2b19=0,\
r2b20=0,\
r2b21=0,\
r2b22=0

#define DEF_R3(n) regtype \
r3b01=expand(1&(n>>0)),\
r3b02=expand(1&(n>>1)),\
r3b03=expand(1&(n>>2)),\
r3b04=expand(1&(n>>3)),\
r3b05=expand(1&(n>>4)),\
r3b06=expand(1&(n>>5)),\
r3b07=expand(1&(n>>6)),\
r3b08=expand(1&(n>>7)),\
r3b09=expand(1&(n>>8)),\
r3b10=expand(1&(n>>9)),\
r3b11=expand(1&(n>>10)),\
r3b12=expand(1&(n>>11)),\
r3b13=expand(1&(n>>12)),\
r3b14=expand(1&(n>>13)),\
r3b15=expand(1&(n>>14)),\
r3b16=expand(1&(n>>15)),\
r3b17=expand(1&(n>>16)),\
r3b18=expand(1&(n>>17)),\
r3b19=expand(1&(n>>18)),\
r3b20=expand(1&(n>>19)),\
r3b21=expand(1&(n>>20)),\
r3b22=expand(1&(n>>21)),\
r3b23=expand(1&(n>>22))

__device__ inline regtype bitselect(regtype a, regtype b, regtype c) {
#if (__CUDA_ARCH__ >= 500)
	regtype out;
	asm("lop3.b32 %0, %1, %2, %3, 0xe4;" : "=r"(out) : "r"(a), "r"(b), "r"(c));
	return out;
#else
	return ((a&c)|(b&(~c)));
#endif
}
//__device__ inline regtype bitselect(regtype a, regtype b, regtype c) {return (a ^ ((c ^ a) & b));}

regtype bitselect_cpu(regtype a, regtype b, regtype c) { return ((a&c) | (b&(~c))); }
inline bool getbit_cpu(char* data, unsigned bnum)
{
	return (bool)(data[bnum >> 3] & (1 << (7 - (bnum & 0x7))));
	//return (bool)(data[bnum]);
}
__device__ inline void ClockR1(DEF_REGS_R1, regtype& f)
{

	// New bit
	regtype nv = b19 ^ b18 ^ b17 ^ b14;
	b19 = bitselect(b18, b19, f);
	b18 = bitselect(b17, b18, f);
	b17 = bitselect(b16, b17, f);
	b16 = bitselect(b15, b16, f);
	b15 = bitselect(b14, b15, f);
	b14 = bitselect(b13, b14, f);
	b13 = bitselect(b12, b13, f);
	b12 = bitselect(b11, b12, f);
	b11 = bitselect(b10, b11, f);
	b10 = bitselect(b09, b10, f);
	b09 = bitselect(b08, b09, f);
	b08 = bitselect(b07, b08, f);
	b07 = bitselect(b06, b07, f);
	b06 = bitselect(b05, b06, f);
	b05 = bitselect(b04, b05, f);
	b04 = bitselect(b03, b04, f);
	b03 = bitselect(b02, b03, f);
	b02 = bitselect(b01, b02, f);
	b01 = bitselect(nv, b01, f);

}

__device__ inline void ClockR2(DEF_REGS_R2, regtype& f)
{
	// New bit
	regtype nv = b22 ^ b21;
	b22 = bitselect(b21, b22, f);
	b21 = bitselect(b20, b21, f);
	b20 = bitselect(b19, b20, f);
	b19 = bitselect(b18, b19, f);
	b18 = bitselect(b17, b18, f);
	b17 = bitselect(b16, b17, f);
	b16 = bitselect(b15, b16, f);
	b15 = bitselect(b14, b15, f);
	b14 = bitselect(b13, b14, f);
	b13 = bitselect(b12, b13, f);
	b12 = bitselect(b11, b12, f);
	b11 = bitselect(b10, b11, f);
	b10 = bitselect(b09, b10, f);
	b09 = bitselect(b08, b09, f);
	b08 = bitselect(b07, b08, f);
	b07 = bitselect(b06, b07, f);
	b06 = bitselect(b05, b06, f);
	b05 = bitselect(b04, b05, f);
	b04 = bitselect(b03, b04, f);
	b03 = bitselect(b02, b03, f);
	b02 = bitselect(b01, b02, f);
	b01 = bitselect(nv, b01, f);
}

__device__ inline void ClockR3(DEF_REGS_R3, regtype& f)
{
	// New bit
	regtype nv = b23 ^ b22 ^ b21 ^ b08;
	b23 = bitselect(b22, b23, f);
	b22 = bitselect(b21, b22, f);
	b21 = bitselect(b20, b21, f);
	b20 = bitselect(b19, b20, f);
	b19 = bitselect(b18, b19, f);
	b18 = bitselect(b17, b18, f);
	b17 = bitselect(b16, b17, f);
	b16 = bitselect(b15, b16, f);
	b15 = bitselect(b14, b15, f);
	b14 = bitselect(b13, b14, f);
	b13 = bitselect(b12, b13, f);
	b12 = bitselect(b11, b12, f);
	b11 = bitselect(b10, b11, f);
	b10 = bitselect(b09, b10, f);
	b09 = bitselect(b08, b09, f);
	b08 = bitselect(b07, b08, f);
	b07 = bitselect(b06, b07, f);
	b06 = bitselect(b05, b06, f);
	b05 = bitselect(b04, b05, f);
	b04 = bitselect(b03, b04, f);
	b03 = bitselect(b02, b03, f);
	b02 = bitselect(b01, b02, f);
	b01 = bitselect(nv, b01, f);

}


__device__ inline void AssertR3Nozeros(DEF_REGS_R3)
{
	assert(0xffffffff == (b01 | b02 | b03 | b04 | b05 | b06 | b07 | b08 | b09 | b10 | b11 | b12 | b13 | b14 | b15 | b16 | b17 | b18 | b19 | b20 | b21 | b22 | b23));
}

__device__ inline void AssertR2Nozeros(DEF_REGS_R2)
{
	assert(0xffffffff == (b01 | b02 | b03 | b04 | b05 | b06 | b07 | b08 | b09 | b10 | b11 | b12 | b13 | b14 | b15 | b16 | b17 | b18 | b19 | b20 | b21 | b22));
}
__device__ inline void AssertR1Nozeros(DEF_REGS_R1)
{
	assert(0xffffffff == (b01 | b02 | b03 | b04 | b05 | b06 | b07 | b08 | b09 | b10 | b11 | b12 | b13 | b14 | b15 | b16 | b17 | b18 | b19));
}

__device__ inline regtype AddAndCheckBitwise(regtype& b0, regtype& b1, regtype& b2, regtype& b3, regtype in)
{
	const int VALUE = 11;
	const regtype y0 = expand(1 & (VALUE >> 0));
	const regtype y1 = expand(1 & (VALUE >> 1));
	const regtype y2 = expand(1 & (VALUE >> 2));
	const regtype y3 = expand(1 & (VALUE >> 3));

	regtype c0 = 0, c1 = 0, c2 = 0, c3 = 0;
	//Check value bitwise if we need to sum futher
	c0 = in & ((b0^y0) | (b1^y1) | (b2^y2) | (b3^y3));

	// Add bitwise
	c1 = b0 & c0; //carry bit 0->1
	c2 = b1 & c1; //carry bit 1->2
	c3 = b2 & c2; //carry bit 2->3
	b0 ^= c0; // assign bit it's new value
	b1 ^= c1;
	b2 ^= c2;
	b3 ^= c3;

	//		printf ("%x ", (b0^y0) | (b1^y1) | (b2^y2) | (b3^y3));
	//Check value again
	return (in & ((b0^y0) | (b1^y1) | (b2^y2) | (b3^y3)));
}

__device__ inline regtype CheckBitwise(regtype& b0, regtype& b1, regtype& b2, regtype& b3)
{
	const int VALUE = 11;
	const regtype y0 = expand(1 & (VALUE >> 0));
	const regtype y1 = expand(1 & (VALUE >> 1));
	const regtype y2 = expand(1 & (VALUE >> 2));
	const regtype y3 = expand(1 & (VALUE >> 3));
	return ((b0^y0) | (b1^y1) | (b2^y2) | (b3^y3));
}

__device__ inline unsigned int bslc2uint(int i)
{
	unsigned int out = 0x0;

	out |=
		((0x0000ffff>>i)&1)<<0 | 
		((0x00ff00ff>>i)&1)<<1 |
		((0xf0f0f0f0>>i)&1)<<2 |
		((0x33333333>>i)&1)<<3 |
		((0x55555555>>i)&1)<<4 ;
	return out;
}

__global__ void kernel_gpu(regtype* stream_expanded_gpu, unsigned rnum)
{

	assert(0x1 == bitselect(0x1, 0x0, 0x1));
	assert(0x0 == bitselect(0x1, 0x0, 0x0));
	assert(0x0 == bitselect(0x0, 0x1, 0x1));
	assert(0x1 == bitselect(0x0, 0x1, 0x0));
	assert(0x0 == bitselect(0x0, 0x0, 0x0));
	assert(0x1 == bitselect(0x1, 0x1, 0x1));
	assert(0x0 == bitselect(0x0, 0x0, 0x1));
	assert(0x1 == bitselect(0x1, 0x1, 0x0));
	if (((blockIdx.x << BLOCKSIZE) + threadIdx.x) == 0) return;
	const unsigned int tid = (blockIdx.x << BLOCKSIZE) + threadIdx.x;
	DEF_R1(rnum);
	DEF_R2((rnum>>19) << 5);
	DEF_R3(tid);

	// STAGE 1 - compute R2 register
	unsigned dead_lanes = 0x0;
	regtype s0 = 0, s1 = 0, s2 = 0, s3 = 0;

	regtype cval_old = r2b22 = r1b19 ^ stream_expanded_gpu[0] ^ r3b23;

	unsigned long long tmp = 0; 
	unsigned long long tmp2 = 0; 
	for (int i = 0; i<64; ++i) {
		// majority
		regtype maj = ~(r1b10 ^ r2b11 ^ r3b11);

		//AssertR1Nozeros(REGS_R1);
		regtype m1 = r1b10 ^ maj;
		ClockR1(REGS_R1, m1);

		//AssertR3Nozeros(REGS_R3);
		regtype m3 = r3b11 ^ maj;
		ClockR3(REGS_R3, m3);

		regtype cval_next = r1b19 ^ stream_expanded_gpu[i] ^ r3b23;

		//AssertR2Nozeros(REGS_R2);
		regtype m2 = r2b11 ^ maj;
		regtype st1mode = AddAndCheckBitwise(s0, s1, s2, s3, m2); // Stage 1 or 2 flag
		dead_lanes |= st1mode & (~m2 & (cval_old ^ cval_next)); // Early conflict!
		r2b21 = bitselect(cval_next, r2b21, st1mode);
		ClockR2(REGS_R2, m2);
		cval_old = cval_next;

		regtype o = r1b19 ^ r2b22 ^ r3b23;
		/*
		if (tid == 0x123)
		{
			tmp |= (unsigned long long)((o & 1))<<i;
			tmp2 |= (unsigned long long)((stream_expanded_gpu[i] & 1))<<i;
		}
		*/

		dead_lanes |= (stream_expanded_gpu[i] ^ o)&(~st1mode); // only for stage 2
		//printf ("%x:", dead_lanes);
		if (dead_lanes == 0xffffffff )
			break;
	}

	if (dead_lanes != 0xffffffff)
	{
		for (int i=0; i<32; ++i)
			if(((dead_lanes >> i)&1) == 0)
			{
				gpu_result =
				(((unsigned)bslc2uint(i)) << 23)|
				((unsigned)tid);
				//if (tid == 0x123) printf ("\n gpures %lx", gpu_result);
				break;
			}
	}

	/*
	if (tid == 0x123)
	{

		printf ("\n %lx", gpu_result);
		printf ("\n %llx", tmp);
		printf ("\n %llx", tmp2);
	}
	*/
	//printf("\n");
	
}

unsigned long long RunKernel(unsigned long long c_stream, unsigned long long start, unsigned long long stop, int device)
{
	//test key 53 bit 0x948000ad800123
	//test stream 0xb3f4c70cdd61313f

	regtype stream_expanded[8*sizeof(c_stream)];
	for (int i = 0; i<8*sizeof(c_stream); ++i)
	{
		stream_expanded[i] = ((c_stream >> i)&1 ? 0xffffffff : 0x0);
		//printf("%i", stream_expanded[i]&1);
	}

	hipDeviceProp_t dp;
	hipGetDeviceProperties(&dp, device);
	//printf(" %i %i ",dp.major, dp.minor);
	if (dp.major < 2)
	{
		printf("\nGPU compute capability too low!");
		exit(1);
	}
	hipSetDevice(device);

	regtype* stream_expanded_gpu;
	m_CopyToGPU(stream_expanded_gpu, stream_expanded, KSTREAMLEN * 2 * 8 * sizeof(regtype));
	assert(expand_cpu(1) == 0xffffffff);

	//  R1 - 19 bits | R2 - 22 bits                                          | R3 - 23 bits
	//  R1 - 19 bits | 11 anderson bits | 6 bits - R2 bits | 5 bitslice bits | R3 - 23 bits 
	//  ^ start      |                  | ^ start          | ^  processed in kernel  ^
	unsigned start_regs = start>>(23+5);
	unsigned stop_regs = stop>>(23+5);

	
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernel_gpu), hipFuncCachePreferL1);
	hipEvent_t kernel_start, kernel_stop;
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_stop);
	hipEventRecord(kernel_start);
	//printf("start-stop %x %x", start_regs, stop_regs);
	unsigned h_result = 0;
	unsigned long long out = 0;
	for (unsigned i = start_regs; i < stop_regs; i++) 
	{
		kernel_gpu <<<(1 << (R3LEN - BLOCKSIZE)), 1<<BLOCKSIZE >>> (stream_expanded_gpu, i);
		#ifndef NDEBUG
		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();
		checkCudaErrors(err);
		#endif
		checkCudaErrors(hipMemcpyFromSymbol(&h_result, HIP_SYMBOL(gpu_result), sizeof(gpu_result), 0, hipMemcpyDeviceToHost));
		if (h_result != 0)
		{
			out =(((unsigned long long)i) << (23+5)) | ((unsigned long long)h_result) ;
			break;
		}
	}
	//printf("\nResult: %llx", out);
	hipDeviceSynchronize();
	hipEventRecord(kernel_stop);
	hipEventSynchronize(kernel_stop);
	float preprocessTime;
	hipEventElapsedTime(&preprocessTime, kernel_start, kernel_stop);


	float tm = preprocessTime / 1000;

	return out;
}
